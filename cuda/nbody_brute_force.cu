#include "hip/hip_runtime.h"
/*
** nbody_brute_force.c - nbody simulation using the brute-force algorithm (O(n*n))
**
**/

#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <math.h>
#include <sys/time.h>
#include <assert.h>
#include <unistd.h>

#ifdef DISPLAY
#include <X11/Xlib.h>
#include <X11/Xutil.h>
#endif

#include "ui.h"
#include "nbody.h"
#include "nbody_tools.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// /usr/local/cuda/samples/common/inc
#include <hip/hip_runtime_api.h> 
FILE* f_out=NULL;

int nparticles=10;      /* number of particles */
float T_FINAL=1.0;     /* simulation end time */
particle_t*particles;

double sum_speed_sq = 0;
double max_acc = 0;
double max_speed = 0;

// Helper to atomicMax on doubles
__device__ static double atomicMax(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(fmaxf(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    return __longlong_as_double(old);
}


void init() {
  /* Nothing to do */
}

#ifdef DISPLAY
Display *theDisplay;  /* These three variables are required to open the */
GC theGC;             /* particle plotting window.  They are externally */
Window theMain;       /* declared in ui.h but are also required here.   */
#endif

/* compute the force that a particle with position (x_pos, y_pos) and mass 'mass'
 * applies to particle p
 */
void compute_force(particle_t*p, double x_pos, double y_pos, double mass) {
  double x_sep, y_sep, dist_sq, grav_base;

  x_sep = x_pos - p->x_pos;
  y_sep = y_pos - p->y_pos;
  dist_sq = MAX((x_sep*x_sep) + (y_sep*y_sep), 0.01);

  /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
  grav_base = GRAV_CONSTANT*(p->mass)*(mass)/dist_sq;

  p->x_force += grav_base*x_sep;
  p->y_force += grav_base*y_sep;
}

/* compute the new position/velocity */
void move_particle(particle_t*p, double step) {

  p->x_pos += (p->x_vel)*step;
  p->y_pos += (p->y_vel)*step;
  double x_acc = p->x_force/p->mass;
  double y_acc = p->y_force/p->mass;
  p->x_vel += x_acc*step;
  p->y_vel += y_acc*step;

  /* compute statistics */
  double cur_acc = (x_acc*x_acc + y_acc*y_acc);
  cur_acc = sqrt(cur_acc);
  double speed_sq = (p->x_vel)*(p->x_vel) + (p->y_vel)*(p->y_vel);
  double cur_speed = sqrt(speed_sq);

  sum_speed_sq += speed_sq;
  max_acc = MAX(max_acc, cur_acc);
  max_speed = MAX(max_speed, cur_speed);
}


/*
  Move particles one time step.

  Update positions, velocity, and acceleration.
  Return local computations.
*/
void all_move_particles(double step)
{
  /* First calculate force for particles. */
  int i;
  for(i=0; i<nparticles; i++) {
    int j;
    particles[i].x_force = 0;
    particles[i].y_force = 0;
    for(j=0; j<nparticles; j++) {
      particle_t*p = &particles[j];
      /* compute the force of particle j on particle i */
      compute_force(&particles[i], p->x_pos, p->y_pos, p->mass);
    }
  }

  /* then move all particles and return statistics */
  for(i=0; i<nparticles; i++) {
    move_particle(&particles[i], step);
  }
}

/* display all the particles */
void draw_all_particles() {
  int i;
  for(i=0; i<nparticles; i++) {
    int x = POS_TO_SCREEN(particles[i].x_pos);
    int y = POS_TO_SCREEN(particles[i].y_pos);
    draw_point (x,y);
  }
}

void print_all_particles(FILE* f) {
  int i;
  for(i=0; i<nparticles; i++) {
    particle_t*p = &particles[i];
    fprintf(f, "particle={pos=(%f,%f), vel=(%f,%f)}\n", p->x_pos, p->y_pos, p->x_vel, p->y_vel);
  }
}

// __global__ void compute_force_atomic(particle_t*p, double x_pos, double y_pos, double mass) {
//   double x_sep, y_sep, dist_sq, grav_base;

//   x_sep = x_pos - p->x_pos;
//   y_sep = y_pos - p->y_pos;
//   dist_sq = MAX((x_sep*x_sep) + (y_sep*y_sep), 0.01);

//   /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
//   grav_base = GRAV_CONSTANT*(p->mass)*(mass)/dist_sq;

//   atomicAdd(&(p->x_force), grav_base*x_sep);
//   atomicAdd(&(p->y_force), grav_base*y_sep);
// }

// __global__ void move_particle_atomic(particle_t*p, double step) {

//   p->x_pos += (p->x_vel)*step;
//   p->y_pos += (p->y_vel)*step;
//   double x_acc = p->x_force/p->mass;
//   double y_acc = p->y_force/p->mass;
//   p->x_vel += x_acc*step;
//   p->y_vel += y_acc*step;

//   /* compute statistics */
//   double cur_acc = (x_acc*x_acc + y_acc*y_acc);
//   cur_acc = sqrt(cur_acc);
//   double speed_sq = (p->x_vel)*(p->x_vel) + (p->y_vel)*(p->y_vel);
//   double cur_speed = sqrt(speed_sq);

//   atomicAdd(&sum_speed_sq, speed_sq);
//   atomicMax(&max_acc, cur_acc);
//   atomicMax(&max_speed, cur_speed);
// }

__global__ void kernel(void) {}

__global__ void reset_forces(particle_t* gpu_particles) {
  int i = blockIdx.x;
  gpu_particles[i].x_force = 0;
  gpu_particles[i].y_force = 0;
}

__global__ void calculate_forces(particle_t* gpu_particles) {
  int i = blockIdx.x;
  int j = blockIdx.y;
  particle_t* p = &gpu_particles[i];
  particle_t* p_distant = &gpu_particles[j];

  double x_sep, y_sep, dist_sq, grav_base;

  x_sep = p->x_pos - p_distant->x_pos;
  y_sep = p->y_pos - p_distant->y_pos;
  dist_sq = MAX((x_sep*x_sep) + (y_sep*y_sep), 0.01);

  /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
  grav_base = GRAV_CONSTANT*(p->mass)*(p_distant->mass)/dist_sq;

  atomicAdd(&(p->x_force), grav_base*x_sep);
  atomicAdd(&(p->y_force), grav_base*y_sep);
}

__global__ void move_all_particles(particle_t* gpu_particles, double step) {
  int i = blockIdx.x;

  particle_t* p = &gpu_particles[i];
  p->x_pos += (p->x_vel)*step;
  p->y_pos += (p->y_vel)*step;
  double x_acc = p->x_force/p->mass;
  double y_acc = p->y_force/p->mass;
  p->x_vel += x_acc*step;
  p->y_vel += y_acc*step;

  /* compute statistics */
  double cur_acc = (x_acc*x_acc + y_acc*y_acc);
  cur_acc = sqrt(cur_acc);
  double speed_sq = (p->x_vel)*(p->x_vel) + (p->y_vel)*(p->y_vel);
  double cur_speed = sqrt(speed_sq);

  atomicAdd(&sum_speed_sq, speed_sq);
  atomicMax(&max_acc, cur_acc);
  atomicMax(&max_speed, cur_speed);
}

// Les kernel sont des points de synchro askip donc ca devrait etre bon
void all_move_particles_kernel(double step, particle_t* gpu_particles) {
  reset_forces <<< nparticles, nparticles >>> (gpu_particles);
  calculate_forces <<< nparticles, nparticles >>> (gpu_particles);
  move_all_particles <<< nparticles, nparticles >>> (gpu_particles, step);
} 


void run_simulation() {
  // CUDA Setup
  particle_t* gpu_particles;
  double gpu_sum_speed_sq, gpu_max_acc, gpu_max_speed;
  size_t size = nparticles * sizeof(particle_t);

  hipMalloc(&gpu_sum_speed_sq, sizeof(double));
  hipMemcpy(&gpu_sum_speed_sq, &sum_speed_sq, sizeof(double), hipMemcpyHostToDevice);
  hipMalloc(&gpu_max_acc, sizeof(double));
  hipMemcpy(&gpu_max_acc, &max_acc, sizeof(double), hipMemcpyHostToDevice);
  hipMalloc(&gpu_max_speed, sizeof(double));
  hipMemcpy(&gpu_max_speed, &max_speed, sizeof(double), hipMemcpyHostToDevice);

  hipMalloc((void**)&gpu_particles, size);
  hipMemcpy(gpu_particles, particles, size, hipMemcpyHostToDevice);

  dim3 grid(nparticles, nparticles);


  double t = 0.0, dt = 0.01;
  while (t < T_FINAL && nparticles>0) {
    /* Update time. */
    t += dt;
    /* Move particles with the current and compute rms velocity. */
    all_move_particles_kernel(dt, gpu_particles);

    /* Adjust dt based on maximum speed and acceleration--this
       simple rule tries to insure that no velocity will change
       by more than 10% */

    dt = 0.1*max_speed/max_acc;

    /* Plot the movement of the particle */
#if DISPLAY
    clear_display();
    draw_all_particles();
    flush_display();
#endif
  }
  
  hipMemcpy(particles, gpu_particles, size, hipMemcpyDeviceToHost);
  hipFree(gpu_particles);
  hipFree(gpu_sum_speed_sq);
  hipFree(gpu_max_acc);
  hipFree(gpu_max_speed);
}

/*
  Simulate the movement of nparticles particles.
*/
int main(int argc, char**argv)
{
  if(argc >= 2) {
    nparticles = atoi(argv[1]);
  }
  if(argc == 3) {
    T_FINAL = atof(argv[2]);
  }

  init();

  /* Allocate global shared arrays for the particles data set. */
  particles = (particle_t*)malloc(sizeof(particle_t)*nparticles);
  all_init_particles(nparticles, particles);

  /* Initialize thread data structures */
#ifdef DISPLAY
  /* Open an X window to display the particles */
  simple_init (100,100,DISPLAY_SIZE, DISPLAY_SIZE);
#endif

  struct timeval t1, t2;
  gettimeofday(&t1, NULL);

  /* Main thread starts simulation ... */
  run_simulation();

  gettimeofday(&t2, NULL);

  double duration = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);

#ifdef DUMP_RESULT
  FILE* f_out = fopen("particles.log", "w");
  assert(f_out);
  print_all_particles(f_out);
  fclose(f_out);
#endif

  printf("-----------------------------\n");
  printf("nparticles: %d\n", nparticles);
  printf("T_FINAL: %f\n", T_FINAL);
  printf("-----------------------------\n");
  printf("Simulation took %lf s to complete\n", duration);

#ifdef DISPLAY
  clear_display();
  draw_all_particles();
  flush_display();

  printf("Hit return to close the window.");

  getchar();
  /* Close the X window used to display the particles */
  XCloseDisplay(theDisplay);
#endif
  return 0;
}
